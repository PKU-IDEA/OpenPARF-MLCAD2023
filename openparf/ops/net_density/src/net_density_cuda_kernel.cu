#include "hip/hip_runtime.h"
/**
 * @file net_density_cuda_kernel.cu
 * @author Yifan Chen (chenyifan2019@pku.edu.cn)
 * @brief 
 * @version 0.1
 * @date 2023-08-19
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include <hip/hip_runtime.h>

#include "ops/net_density/src/risa_parameters.h"
#include "util/atomic_ops.cuh"
#include "util/limits.cuh"
#include "util/util.cuh"

OPENPARF_BEGIN_NAMESPACE

template<typename T>
inline __device__ DEFINE_NET_WIRING_DISTRIBUTION_MAP_WEIGHT;

template<typename T, typename U, typename AtomicOp>
__global__ void NetDensityCudaKernel(T *pin_pos,
        int32_t                  *netpin_start,
        int32_t                  *flat_netpin,
        T                        *net_weights,
        T                         bin_size_x,
        T                         bin_size_y,
        T                         xl,
        T                         yl,
        T                         xh,
        T                         yh,
        int32_t                   num_bins_x,
        int32_t                   num_bins_y,
        int32_t                   num_nets,
        AtomicOp                  atomic_op,
        U                        *horizontal_utilization_map,
        U                        *vertical_utilization_map) {
  int32_t i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < num_nets) {
    int32_t start = netpin_start[i];
    int32_t end   = netpin_start[i + 1];

    T       x_max = -cuda::numeric_limits<T>::max();
    T       x_min = cuda::numeric_limits<T>::max();
    T       y_max = -cuda::numeric_limits<T>::max();
    T       y_min = cuda::numeric_limits<T>::max();

    for (int32_t j = start; j < end; ++j) {
      int32_t pin_id = flat_netpin[j];
      T       xx     = pin_pos[pin_id << 1];
      T       yy     = pin_pos[(pin_id << 1) | 1];
      x_max          = OPENPARF_STD_NAMESPACE::max(xx, x_max);
      x_min          = OPENPARF_STD_NAMESPACE::min(xx, x_min);
      y_max          = OPENPARF_STD_NAMESPACE::max(yy, y_max);
      y_min          = OPENPARF_STD_NAMESPACE::min(yy, y_min);
    }

    // compute the bin box that this net will affect
    auto bin_index_xl = OPENPARF_STD_NAMESPACE::floor((x_min - xl) / bin_size_x);
    auto bin_index_xh = OPENPARF_STD_NAMESPACE::ceil((x_max - xl) / bin_size_x);
    auto bin_index_yl = OPENPARF_STD_NAMESPACE::floor((y_min - yl) / bin_size_y);
    auto bin_index_yh = OPENPARF_STD_NAMESPACE::ceil((y_max - yl) / bin_size_y);

    bin_index_xl      = OPENPARF_STD_NAMESPACE::max(bin_index_xl, (decltype(bin_index_xl)) 0);
    bin_index_xh      = OPENPARF_STD_NAMESPACE::min(bin_index_xh, (decltype(bin_index_xh)) num_bins_x);
    bin_index_yl      = OPENPARF_STD_NAMESPACE::max(bin_index_yl, (decltype(bin_index_yl)) 0);
    bin_index_yh      = OPENPARF_STD_NAMESPACE::min(bin_index_yh, (decltype(bin_index_yh)) num_bins_y);

    /**
      * Follow Wuxi's implementation, a tolerance is added to avoid
      * 0-size bounding box
      */
    T wt_h              = 1. / (bin_index_yh - bin_index_yl + cuda::numeric_limits<T>::epsilon());
    T wt_v              = 1. / (bin_index_xh - bin_index_xl + cuda::numeric_limits<T>::epsilon());

    if (net_weights) {
      wt_h *= net_weights[i];
      wt_v *= net_weights[i];
    }

    for (int32_t x = bin_index_xl; x < bin_index_xh; ++x) {
      for (int32_t y = bin_index_yl; y < bin_index_yh; ++y) {
        int32_t index = x * num_bins_y + y;
        atomic_op(horizontal_utilization_map + index, wt_h);
        atomic_op(vertical_utilization_map + index, wt_v);
      }
    }
  }
}

template<typename T>
void NetDensityCudaLauncher(T *pin_pos,
        int32_t         *netpin_start,
        int32_t         *flat_netpin,
        T               *net_weights,
        T                bin_size_x,
        T                bin_size_y,
        T                xl,
        T                yl,
        T                xh,
        T                yh,
        int32_t          num_bins_x,
        int32_t          num_bins_y,
        int32_t          num_nets,
        int32_t          deterministic_flag,
        T               *horizontal_utilization_map,
        T               *vertical_utilization_map) {
  if (deterministic_flag) {
    using AtomicIntType                   = unsigned long long int;
    AtomicIntType            scale_factor = 1e10;
    AtomicAdd<AtomicIntType> atomic_op(scale_factor);
    int32_t                  thread_count = 256;
    int32_t                  block_count  = ceilDiv(num_nets, thread_count);
    AtomicIntType           *buf_hmap     = nullptr;
    AtomicIntType           *buf_vmap     = nullptr;
    allocateCUDA(buf_hmap, num_bins_y * num_bins_x);
    allocateCUDA(buf_vmap, num_bins_y * num_bins_x);
    DEFER({
      destroyCUDA(buf_hmap);
      destroyCUDA(buf_vmap);
    });
    copyScaleArray<<<block_count, thread_count>>>(buf_hmap, horizontal_utilization_map, scale_factor,
            num_bins_y * num_bins_x);
    copyScaleArray<<<block_count, thread_count>>>(buf_vmap, vertical_utilization_map, scale_factor,
            num_bins_y * num_bins_x);

    NetDensityCudaKernel<T, AtomicIntType, decltype(atomic_op)>
            <<<(uint32_t) block_count, {(uint32_t) thread_count, 1u, 1u}>>>(pin_pos, netpin_start, flat_netpin,
                    net_weights, bin_size_x, bin_size_y, xl, yl, xh, yh, num_bins_x, num_bins_y, num_nets, atomic_op,
                    buf_hmap, buf_vmap);

    copyScaleArray<<<block_count, thread_count>>>(horizontal_utilization_map, buf_hmap,
            static_cast<T>(1.0 / scale_factor), num_bins_y * num_bins_x);
    copyScaleArray<<<block_count, thread_count>>>(vertical_utilization_map, buf_vmap,
            static_cast<T>(1.0 / scale_factor), num_bins_y * num_bins_x);
  } else {
    AtomicAdd<T> atomic_op;
    int32_t      thread_count = 256;
    int32_t      block_count  = ceilDiv(num_nets, thread_count);
    NetDensityCudaKernel<T, T, decltype(atomic_op)><<<(uint32_t) block_count, {(uint32_t) thread_count, 1u, 1u}>>>(pin_pos,
            netpin_start, flat_netpin, net_weights, bin_size_x, bin_size_y, xl, yl, xh, yh, num_bins_x, num_bins_y,
            num_nets, atomic_op, horizontal_utilization_map, vertical_utilization_map);
  }
}

// manually instantiate the template function
#define REGISTER_KERNEL_LAUNCHER(T)                                                                                    \
  template void NetDensityCudaLauncher<T>(T * pin_pos, int32_t * netpin_start, int32_t * flat_netpin, T * net_weights,       \
          T bin_size_x, T bin_size_y, T xl, T yl, T xh, T yh, int32_t num_bins_x, int32_t num_bins_y,                  \
          int32_t num_nets, int32_t deterministic_flag, T * horizontal_utilization_map, T * vertical_utilization_map);

REGISTER_KERNEL_LAUNCHER(float)
REGISTER_KERNEL_LAUNCHER(double)

#undef REGISTER_KERNEL_LAUNCHER

OPENPARF_END_NAMESPACE
